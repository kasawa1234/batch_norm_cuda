#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

#include "constants.h"


// ———————————————————————————————————————————————————————————————
/*                          Conv Forward                          */
// ———————————————————————————————————————————————————————————————


template <typename scalar_t>
__global__ void mean_conv_kernel(
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> input_data,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> output_data
){
    // declare a shared memory space as same as one block
    __shared__ scalar_t shared_memory[BLOCK_SIZE_BATCH][BLOCK_SIZE_FEATURE];
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int c = blockIdx.y * blockDim.y + threadIdx.y;
    const int thread_id_n = threadIdx.x;
    const int thread_id_c = threadIdx.y;
    const int h = input_data.size(2);   // height
    const int w = input_data.size(3);   // width

    
    // if the loc cover our data, load in shared memory
    if (n < input_data.size(0) && c < input_data.size(1)){
        shared_memory[thread_id_n][thread_id_c] = static_cast<scalar_t>(0);
        for(int i = 0 ; i < h; i++){
            for(int j = 0; j < w; j++){
                shared_memory[thread_id_n][thread_id_c] += input_data[n][c][i][j];
            }
        }
    } else {
        shared_memory[thread_id_n][thread_id_c] = static_cast<scalar_t>(0);
    }
    __syncthreads();   
    
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
        if (thread_id_n < offset) {
            shared_memory[thread_id_n][thread_id_c] += shared_memory[thread_id_n + offset][thread_id_c];
        }
        __syncthreads();        // wait, till all threads in this block reach
    }
    
    // after this for loop, all should be set, so dump the data and calculate the mean
    if (thread_id_n == 0) {
        output_data[c] = shared_memory[0][thread_id_c] / static_cast<scalar_t>(input_data.size(0) * h * w);
    }
}

template <typename scalar_t>
__global__ void std_conv_kernel(
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> input_data,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> mean,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> batch_norm_output
){
    const int N = input_data.size(0);
    const int h = input_data.size(2);   //height
    const int w = input_data.size(3);   //width

    // declare a shared memory space as same as one block
    __shared__ scalar_t shared_memory[BLOCK_SIZE_BATCH][BLOCK_SIZE_FEATURE];
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int c = blockIdx.y * blockDim.y + threadIdx.y;
    const int thread_id_n = threadIdx.x;
    const int thread_id_c = threadIdx.y;


    // if the loc cover our data, load in shared memory
    if (n < input_data.size(0) && c < input_data.size(1)){
        shared_memory[thread_id_n][thread_id_c] = static_cast<scalar_t>(0);
        for(int i = 0 ; i < h; i++){
            for(int j = 0; j < w; j++){
                shared_memory[thread_id_n][thread_id_c] += (input_data[n][c][i][j] - mean[c]) * (input_data[n][c][i][j] - mean[c]);
            }
        }
    } else {
        shared_memory[thread_id_n][thread_id_c] = static_cast<scalar_t>(0);
    }
    __syncthreads();            // need to fully load all items into shared_memory

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
        if (thread_id_n < offset) {
            shared_memory[thread_id_n][thread_id_c] += shared_memory[thread_id_n + offset][thread_id_c];
        }
        __syncthreads();        // wait, till all threads in this block reach
    }
    
    // after this for loop, all should be set, so dump the data and calculate the std
    if (thread_id_n == 0) {
        batch_norm_output[N][c][0][0] = sqrt(shared_memory[0][thread_id_c] / static_cast<scalar_t>(input_data.size(0) * h * w + EPSILON));
    }
}

template <typename scalar_t>
__global__ void bn_forward_conv_kernel(
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> input_data,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> mean,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> gamma,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> beta,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> output_data
){
    // batch size
    const int N = input_data.size(0);
    const int h = input_data.size(2);   //height
    const int w = input_data.size(3);   //width

    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int c = blockIdx.y * blockDim.y + threadIdx.y;

    if (n >= input_data.size(0) || c >= input_data.size(1)) return;

    for(int i = 0 ; i < h; i++){
        for(int j = 0; j < w; j++){
            output_data[n][c][i][j] = gamma[c] * (input_data[n][c][i][j] - mean[c]) / output_data[N][c][0][0] + beta[c];
        }
    }
}

torch::Tensor bn_forward_conv_cuda(
    const torch::Tensor X,
    const torch::Tensor gamma,
    const torch::Tensor beta
){
    // X: (n, c), n is parallel
    const int N = X.size(0);
    const int C = X.size(1);
    const int H = X.size(2);
    const int W = X.size(3);
    // std::cout << N << ", " << C << std::endl;

    torch::Tensor mean = torch::zeros({C}, X.options());

    const dim3 threads_mean(BLOCK_SIZE_BATCH, BLOCK_SIZE_FEATURE);
    const dim3 blocks_mean((N + threads_mean.x - 1) / threads_mean.x, (C + threads_mean.y - 1) / threads_mean.y);

    // std::cout << "blocks mean: " << blocks_mean.x << ", " << blocks_mean.y << std::endl;

    // launch the kernel
    AT_DISPATCH_FLOATING_TYPES(X.type(), "mean_conv_kernel",
    ([&] {
        mean_conv_kernel<scalar_t><<<blocks_mean, threads_mean>>>(
            X.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            mean.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>()
        );
    }));
    
    // calculate std
    // batch_norm_out: bn_forward + std_eps
    torch::Tensor batch_norm_out = torch::zeros({N + 1, C, H, W}, X.options());

    // standard share the same block size with mean
    // std::cout << "blocks std: " << blocks_mean.x << ", " << blocks_mean.y << std::endl;

    // launch the kernel
    AT_DISPATCH_FLOATING_TYPES(X.type(), "std_conv_kernel",
    ([&] {
        std_conv_kernel<scalar_t><<<blocks_mean, threads_mean>>>(
            X.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            mean.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
            batch_norm_out.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>()
        );
    }));

    // batch norm will use a even dispatched block size
    const dim3 threads_batch_norm(BLOCK_SIZE_BN_X, BLOCK_SIZE_BN_Y);
    const dim3 blocks_batch_norm((N + threads_batch_norm.x - 1) / threads_batch_norm.x, (C + threads_batch_norm.y - 1) / threads_batch_norm.y);

    // std::cout << "blocks batch norm: " << blocks_batch_norm.x << ", " << blocks_batch_norm.y << std::endl;

    // launch the kernel
    AT_DISPATCH_FLOATING_TYPES(X.type(), "bn_forward_conv_kernel",
    ([&] {
        bn_forward_conv_kernel<scalar_t><<<blocks_batch_norm, threads_batch_norm>>>(
            X.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            mean.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
            gamma.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
            beta.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
            batch_norm_out.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>()
        );
    }));
    return batch_norm_out;
}

/*                          Conv Backward                          */

template <typename scalar_t>
__global__ void dx_sum_conv_kernel(
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> dL_dout,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> gamma_1d,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> dx_sum
){
    // declare a shared memory space as same as one block
    __shared__ scalar_t shared_memory[BLOCK_SIZE_BATCH][BLOCK_SIZE_FEATURE];
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int c = blockIdx.y * blockDim.y + threadIdx.y;
    const int thread_id_n = threadIdx.x;
    const int thread_id_c = threadIdx.y;

    const int h = dL_dout.size(2);   //height
    const int w = dL_dout.size(3);   //width
    

    // if the loc cover our data, load in shared memory
    if (n < dL_dout.size(0) && c < dL_dout.size(1)){
        shared_memory[thread_id_n][thread_id_c] = static_cast<scalar_t>(0);
        for(int i = 0; i < h; i++){
            for(int j = 0; j < w; j++){
                shared_memory[thread_id_n][thread_id_c] += dL_dout[n][c][i][j] * gamma_1d[c];
            }
        }
    } else {
        shared_memory[thread_id_n][thread_id_c] = static_cast<scalar_t>(0);
    }
    __syncthreads();            // need to fully load all items into shared_memory

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
        if (thread_id_n < offset) {
            shared_memory[thread_id_n][thread_id_c] += shared_memory[thread_id_n + offset][thread_id_c];
        }
        __syncthreads();        // wait, till all threads in this block reach
    }
    
    // after this for loop, all should be set, so dump the data and calculate the mean
    if (thread_id_n == 0) {
        dx_sum[c] = shared_memory[0][thread_id_c];
    }
}

template <typename scalar_t>
__global__ void dx_norm_sum_conv_kernel(
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> dL_dout,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> gamma,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> normalized,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> dx_norm_sum
){
    // declare a shared memory space as same as one block
    __shared__ scalar_t shared_memory[BLOCK_SIZE_BATCH][BLOCK_SIZE_FEATURE];
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int c = blockIdx.y * blockDim.y + threadIdx.y;
    const int thread_id_n = threadIdx.x;
    const int thread_id_c = threadIdx.y;

    const int h = dL_dout.size(2);   //height
    const int w = dL_dout.size(3);   //width

    // if the loc cover our data, load in shared memory
    if (n < dL_dout.size(0) && c < dL_dout.size(1)){
        shared_memory[thread_id_n][thread_id_c] = static_cast<scalar_t>(0);
        for(int i = 0; i < h; i++){
            for(int j = 0; j < w; j++){
                shared_memory[thread_id_n][thread_id_c] += dL_dout[n][c][i][j] * gamma[c] * normalized[n][c][i][j];
            }
        }
    
    } else {
        shared_memory[thread_id_n][thread_id_c] = static_cast<scalar_t>(0);
    }
    __syncthreads();            // need to fully load all items into shared_memory

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
        if (thread_id_n < offset) {
            shared_memory[thread_id_n][thread_id_c] += shared_memory[thread_id_n + offset][thread_id_c];
        }
        __syncthreads();        // wait, till all threads in this block reach
    }
    
    // after this for loop, all should be set, so dump the data and calculate the mean
    if (thread_id_n == 0) {
        dx_norm_sum[c] = shared_memory[0][thread_id_c];
    }
}

template <typename scalar_t>
__global__ void grad_gamma_conv_kernel(
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> dL_dout,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> normalized,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> output
){
    const int N = dL_dout.size(0);

    // declare a shared memory space as same as one block
    __shared__ scalar_t shared_memory[BLOCK_SIZE_BATCH][BLOCK_SIZE_FEATURE];
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int c = blockIdx.y * blockDim.y + threadIdx.y;
    const int thread_id_n = threadIdx.x;
    const int thread_id_c = threadIdx.y;

    const int h = dL_dout.size(2);   // height
    const int w = dL_dout.size(3);   // width

    // if the loc cover our data, load in shared memory
    if (n < dL_dout.size(0) && c < dL_dout.size(1)){
        shared_memory[thread_id_n][thread_id_c] = static_cast<scalar_t>(0);
        for(int i = 0; i < h; i++){
            for(int j = 0; j < w; j++){
                shared_memory[thread_id_n][thread_id_c] += dL_dout[n][c][i][j] * normalized[n][c][i][j];
            }
        }
    } else {
        shared_memory[thread_id_n][thread_id_c] = static_cast<scalar_t>(0);
    }
    __syncthreads();            // need to fully load all items into shared_memory

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
        if (thread_id_n < offset) {
            shared_memory[thread_id_n][thread_id_c] += shared_memory[thread_id_n + offset][thread_id_c];
        }
        __syncthreads();        // wait, till all threads in this block reach
    }
    
    // after this for loop, all should be set, so dump the data and calculate the mean
    if (thread_id_n == 0) {
        output[N][c][0][0] = shared_memory[0][thread_id_c];
    }
}

template <typename scalar_t>
__global__ void grad_beta_conv_kernel(
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> dL_dout,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> output
){
    const int N = dL_dout.size(0);

    // declare a shared memory space as same as one block
    __shared__ scalar_t shared_memory[BLOCK_SIZE_BATCH][BLOCK_SIZE_FEATURE];
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int c = blockIdx.y * blockDim.y + threadIdx.y;
    const int thread_id_n = threadIdx.x;
    const int thread_id_c = threadIdx.y;

    const int h = dL_dout.size(2);   // height
    const int w = dL_dout.size(3);   // width

    // if the loc cover our data, load in shared memory
    if (n < dL_dout.size(0) && c < dL_dout.size(1)){
        shared_memory[thread_id_n][thread_id_c] = static_cast<scalar_t>(0);
        for(int i = 0; i < h; i++){
            for(int j = 0; j < w; j++){
                shared_memory[thread_id_n][thread_id_c] += dL_dout[n][c][i][j];
            }
        }

    } else {
        shared_memory[thread_id_n][thread_id_c] = static_cast<scalar_t>(0);
    }
    __syncthreads();            // need to fully load all items into shared_memory

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
        if (thread_id_n < offset) {
            shared_memory[thread_id_n][thread_id_c] += shared_memory[thread_id_n + offset][thread_id_c];
        }
        __syncthreads();        // wait, till all threads in this block reach
    }
    
    // after this for loop, all should be set, so dump the data and calculate the mean
    if (thread_id_n == 0) {
        output[N + 1][c][0][0] = shared_memory[0][thread_id_c];
    }
}

template <typename scalar_t>
__global__ void bn_backward_input_conv_kernel(
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> dL_dout,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> gamma,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> dx_sum,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> dx_norm_sum,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> normalized,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> std_eps,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> dL_dinput
){
    const int N = normalized.size(0);
    const int h = dL_dout.size(2);   // height  
    const int w = dL_dout.size(3);   // width

    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int c = blockIdx.y * blockDim.y + threadIdx.y;

    if (n >= normalized.size(0) || c >= normalized.size(1)) return;

    for(int i = 0; i < h; i++){
        for(int j = 0; j < w; j++){
            dL_dinput[n][c][i][j] = (N * h * w * dL_dout[n][c][i][j] * gamma[c] - dx_sum[c] - normalized[n][c][i][j] * dx_norm_sum[c]) / (N * h * w * std_eps[c]);
        }
    }
}

torch::Tensor bn_backward_conv_cuda(
    const torch::Tensor dL_dout,
    const torch::Tensor normalized,
    const torch::Tensor gamma,
    const torch::Tensor std_eps
){
    const int N = normalized.size(0);
    const int C = normalized.size(1);
    const int H = normalized.size(2);
    const int W = normalized.size(3);
    // std::cout << N << ", " << C << std::endl;

    torch::Tensor dx_sum = torch::zeros({C}, normalized.options());

    // using the same block size as mean
    const dim3 threads_sum(BLOCK_SIZE_BATCH, BLOCK_SIZE_FEATURE);
    const dim3 blocks_sum((N + threads_sum.x - 1) / threads_sum.x, (C + threads_sum.y - 1) / threads_sum.y);

    // std::cout << "blocks dx_sum: " << blocks_sum.x << ", " << blocks_sum.y << std::endl;

    AT_DISPATCH_FLOATING_TYPES(normalized.type(), "dx_sum_conv_kernel",
    ([&] {
        dx_sum_conv_kernel<scalar_t><<<blocks_sum, threads_sum>>>(
            dL_dout.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            gamma.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
            dx_sum.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>()
        );
    }));

    torch::Tensor dx_norm_sum = torch::zeros({C}, normalized.options());

    // std::cout << "blocks dx_norm_sum: " << blocks_sum.x << ", " << blocks_sum.y << std::endl;

    AT_DISPATCH_FLOATING_TYPES(normalized.type(), "dx_norm_sum_conv_kernel",
    ([&] {
        dx_norm_sum_conv_kernel<scalar_t><<<blocks_sum, threads_sum>>>(
            dL_dout.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            gamma.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
            normalized.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            dx_norm_sum.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>()
        );
    }));

    // bn_backward_output: grad_input + grad_gamma + grad_beta
    torch::Tensor bn_backward_output = torch::zeros({N + 2, C, H, W}, normalized.options());

    // std::cout << "blocks grad_gamma: " << blocks_sum.x << ", " << blocks_sum.y << std::endl;

    AT_DISPATCH_FLOATING_TYPES(normalized.type(), "grad_gamma_conv_kernel",
    ([&] {
        grad_gamma_conv_kernel<scalar_t><<<blocks_sum, threads_sum>>>(
            dL_dout.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            normalized.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            bn_backward_output.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>()
        );
    }));

    // std::cout << "blocks grad_beta: " << blocks_sum.x << ", " << blocks_sum.y << std::endl;

    AT_DISPATCH_FLOATING_TYPES(normalized.type(), "grad_beta_conv_kernel",
    ([&] {
        grad_beta_conv_kernel<scalar_t><<<blocks_sum, threads_sum>>>(
            dL_dout.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            bn_backward_output.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>()
        );
    }));

    // batch norm will use a even dispatched block size
    const dim3 threads_batch_norm(BLOCK_SIZE_BN_X, BLOCK_SIZE_BN_Y);
    const dim3 blocks_batch_norm((N + threads_batch_norm.x - 1) / threads_batch_norm.x, (C + threads_batch_norm.y - 1) / threads_batch_norm.y);

    // std::cout << "blocks batch norm backwards: " << blocks_batch_norm.x << ", " << blocks_batch_norm.y << std::endl;

    AT_DISPATCH_FLOATING_TYPES(normalized.type(), "bn_backward_input_conv_kernel",
    ([&] {
        bn_backward_input_conv_kernel<scalar_t><<<blocks_batch_norm, threads_batch_norm>>>(
            dL_dout.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            gamma.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
            dx_sum.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
            dx_norm_sum.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
            normalized.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            std_eps.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
            bn_backward_output.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>()
        );
    }));

    return bn_backward_output;
}
